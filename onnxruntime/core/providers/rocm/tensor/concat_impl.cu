#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/rocm/cu_inc/common.cuh"
#include "core/providers/rocm/hip_common.h"
#include "concat_impl.h"

namespace onnxruntime {
namespace rocm {

template <typename T>
__global__ void _ConcatKernel(const fast_divmod block_size_including_axis_dim_div,
                              const fast_divmod block_size_inside_axis_dim_div,
                              const int64_t* concat_sizes,
                              const int64_t* concat_sizes_range,
                              const int64_t* axis_dimension_input_output_mapping,
                              T* output_data,
                              const void** input_ptr,
                              const HIP_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  HIP_LONG input_pos = 0;

  int outter_block_index = 0;
  int block_index = 0;
  int offset = 0;

  block_size_including_axis_dim_div.divmod(id, outter_block_index, offset);
  block_size_inside_axis_dim_div.divmod(offset, block_index, offset);

  int input_index = axis_dimension_input_output_mapping[block_index];
  int64_t range_left = (input_index == 0) ? 0 : concat_sizes_range[input_index - 1];
  int block_offset = block_index - range_left;

  input_pos = (outter_block_index * concat_sizes[input_index] + block_offset) *
               block_size_inside_axis_dim_div.d_ +
               offset;

  output_data[id] = reinterpret_cast<const T*>(input_ptr[input_index])[input_pos];
}

Status ConcatImpl(const size_t element_bytes,
                  const int block_size_including_axis_dim,
                  const int block_size_inside_axis_dim,
                  const int64_t* concat_sizes,
                  const int64_t* concat_sizes_range,
                  const int64_t* axis_dimension_input_output_mapping,
                  void* output_data,
                  const void** input_ptr,
                  const size_t N) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));

  fast_divmod block_size_including_axis_dim_div = fast_divmod(block_size_including_axis_dim);
  fast_divmod block_size_inside_axis_dim_div = fast_divmod(block_size_inside_axis_dim);

  switch (element_bytes) {
    case sizeof(int8_t):
      hipLaunchKernelGGL(_ConcatKernel, dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
          block_size_including_axis_dim_div, block_size_inside_axis_dim_div,
          concat_sizes, concat_sizes_range, axis_dimension_input_output_mapping,
          reinterpret_cast<int8_t*>(output_data),
          input_ptr,
          (HIP_LONG)N);
      break;
    case sizeof(int16_t):
      hipLaunchKernelGGL(_ConcatKernel, dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
          block_size_including_axis_dim_div, block_size_inside_axis_dim_div,
          concat_sizes, concat_sizes_range, axis_dimension_input_output_mapping,
          reinterpret_cast<int16_t*>(output_data),
          input_ptr,
          (HIP_LONG)N);
      break;
    case sizeof(int32_t):
      hipLaunchKernelGGL(_ConcatKernel, dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
          block_size_including_axis_dim_div, block_size_inside_axis_dim_div,
          concat_sizes, concat_sizes_range, axis_dimension_input_output_mapping,
          reinterpret_cast<int32_t*>(output_data),
          input_ptr,
          (HIP_LONG)N);
      break;
    case sizeof(int64_t):
      hipLaunchKernelGGL(_ConcatKernel, dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
          block_size_including_axis_dim_div, block_size_inside_axis_dim_div,
          concat_sizes, concat_sizes_range, axis_dimension_input_output_mapping,
          reinterpret_cast<int64_t*>(output_data),
          input_ptr,
          (HIP_LONG)N);
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for Concat operator");
  }

  return Status::OK();
}

}  // namespace rocm
}  // namespace onnxruntime
