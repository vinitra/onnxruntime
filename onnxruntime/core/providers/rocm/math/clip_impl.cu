#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/hip/math/clip_impl.h"
#include "core/providers/hip/cu_inc/common.cuh"

namespace onnxruntime {
namespace rocm {
template <typename T>
__global__ void _Clip(const T* input, T* output, T min, T max, size_t N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  output[id] = (input[id] < min) ? min : ((input[id] > max) ? max : input[id]);
}

template <typename T>
void ClipImpl(const T* input_data, T* output_data, T min, T max, size_t count) {
  typedef typename ToHipType<T>::MappedType HipT;

  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  hipLaunchKernelGGL(HIP_KERNEL_NAME(_Clip<HipT>), dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, reinterpret_cast<const HipT*>(input_data),
                                                                  reinterpret_cast<HipT*>(output_data),
                                                                  *reinterpret_cast<HipT*>(&min),
                                                                  *reinterpret_cast<HipT*>(&max),
                                                                  count);
}

template void ClipImpl<float>(const float* input_data, float* output_data, float min, float max, size_t count);
template void ClipImpl<double>(const double* input_data, double* output_data, double min, double max, size_t count);
template void ClipImpl<MLFloat16>(const MLFloat16* input_data, MLFloat16* output_data, MLFloat16 min, MLFloat16 max, size_t count);
template void ClipImpl<int8_t>(const int8_t* input_data, int8_t* output_data, int8_t min, int8_t max, size_t count);
template void ClipImpl<uint8_t>(const uint8_t* input_data, uint8_t* output_data, uint8_t min, uint8_t max, size_t count);
template void ClipImpl<int64_t>(const int64_t* input_data, int64_t* output_data, int64_t min, int64_t max, size_t count);
template void ClipImpl<uint64_t>(const uint64_t* input_data, uint64_t* output_data, uint64_t min, uint64_t max, size_t count);

}  // namespace rocm
}  // namespace onnxruntime
