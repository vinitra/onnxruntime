#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/hip/cu_inc/common.cuh"
#include "expand_impl.h"
#include "core/providers/hip/hip_common.h"

namespace onnxruntime {
namespace hip {

template <typename T, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void _FillFromDataPtrKernel(T* output_data, const T* input_data, HIP_LONG N) {
  HIP_LONG id = NumElementsPerThread * blockDim.x * blockIdx.x + threadIdx.x;
  T val = *input_data;
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      output_data[id] = val;
      id += NumThreadsPerBlock;
    }
  }
}

template <typename T>
void FillFromDataPtr(T* output_data, const T* input_data, int64_t count) {
  int blocksPerGrid = gsl::narrow_cast<int>(CeilDiv(count, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
  HIP_LONG N = static_cast<HIP_LONG>(count);
  hipLaunchKernelGGL(_FillFromDataPtrKernel<T, GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread>, dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, output_data, input_data, N);
}

template <typename T>
__global__ void ExpandKernel2D(
    const int N,
    const T* input_data,
    T* output_data,
    const fast_divmod fdm_output_stride0,
    const int input_view_stride0,
    const int input_view_stride1) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);

  int dim0, dim1;
  fdm_output_stride0.divmod(id, dim0, dim1);
  output_data[id] = input_data[dim0 * input_view_stride0 + dim1 * input_view_stride1];
}

template <typename T>
__global__ void ExpandKernel(
    const int rank,
    const int N,
    const T* input_data,
    T* output_data,
    const fast_divmod* fdm_output_strides,
    const int64_t* input_view_strides) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);

  int dim, r = id, input_index = 0;
  for (int i = 0; i < rank; ++i) {
    fdm_output_strides[i].divmod(r, dim, r);
    input_index += dim * input_view_strides[i];
  }
  output_data[id] = input_data[input_index];
}

Status ExpandByFill(const size_t element_size, const int N, const void* input_data, void* output_data) {
#define EXPAND_FILL_ON(TYPE)                                   \
  case sizeof(TYPE):                                           \
    FillFromDataPtr(reinterpret_cast<TYPE*>(output_data),      \
                    reinterpret_cast<const TYPE*>(input_data), \
                    static_cast<int64_t>(N));                  \
    break

  switch (element_size) {
    EXPAND_FILL_ON(int8_t);
    EXPAND_FILL_ON(int16_t);
    EXPAND_FILL_ON(int32_t);
    EXPAND_FILL_ON(int64_t);
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for Expand operator");
  }
  return Status::OK();
}

Status Expand2D(
    const size_t element_size,
    const int N,
    const void* input_data,
    void* output_data,
    const fast_divmod fdm_output_stride0,
    const int input_view_stride0,
    const int input_view_stride1) {
#define EXPAND2D_ON(TYPE)                                                                   \
  case sizeof(TYPE):                                                                        \
    hipLaunchKernelGGL(ExpandKernel2D, dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0,                       \
        N, reinterpret_cast<const TYPE*>(input_data), reinterpret_cast<TYPE*>(output_data), \
        fdm_output_stride0, input_view_stride0, input_view_stride1);                        \
    break

  int blocksPerGrid = gsl::narrow_cast<int>(CeilDiv(N, GridDim::maxThreadsPerBlock));
  switch (element_size) {
    EXPAND2D_ON(int8_t);
    EXPAND2D_ON(int16_t);
    EXPAND2D_ON(int32_t);
    EXPAND2D_ON(int64_t);
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for Expand operator");
  }
  return Status::OK();
}

Status ExpandImpl(
    const size_t element_size,
    const int N_output,
    const int N_input,
    const void* input_data,
    void* output_data,
    HipKernel::HipAsyncBuffer<fast_divmod>& fdm_output_strides,
    HipKernel::HipAsyncBuffer<int64_t>& input_view_strides) {
  const int rank = static_cast<int>(fdm_output_strides.count());
  if (rank == 1) {
    if (N_input == N_output) {
      HIP_RETURN_IF_ERROR(hipMemcpyAsync(output_data, input_data, N_output * element_size, hipMemcpyDeviceToDevice));
    } else {  // N_input == 1
      return ExpandByFill(element_size, N_output, input_data, output_data);
    }
  } else if (rank == 2) {
    return Expand2D(element_size, N_output, input_data, output_data,
                    fdm_output_strides.CpuSpan()[0],
                    static_cast<int>(input_view_strides.CpuSpan()[0]),
                    static_cast<int>(input_view_strides.CpuSpan()[1]));
  }

  int blocksPerGrid = gsl::narrow_cast<int>(CeilDiv(N_output, GridDim::maxThreadsPerBlock));
  fdm_output_strides.CopyToGpu();
  input_view_strides.CopyToGpu();

#define EXPAND_ON(TYPE)                                                                                  \
  case sizeof(TYPE):                                                                                     \
    hipLaunchKernelGGL(ExpandKernel, dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0,                                      \
        rank, N_output, reinterpret_cast<const TYPE*>(input_data), reinterpret_cast<TYPE*>(output_data), \
        fdm_output_strides.GpuPtr(), input_view_strides.GpuPtr());                                       \
    break

  switch (element_size) {
    EXPAND_ON(uint8_t);
    EXPAND_ON(uint16_t);
    EXPAND_ON(uint32_t);
    EXPAND_ON(uint64_t);
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for Expand operator");
  }
  return Status::OK();
}

}  // namespace hip
}  // namespace onnxruntime
