#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "orttraining/training_ops/hip/tensor/gather_grad_impl.h"
#include "core/providers/hip/cu_inc/common.cuh"

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/unique.h>
#include <thrust/sort.h>

namespace onnxruntime {
namespace hip {

static const int WARP_SIZE = 32;

template <typename T, typename Tin>
__global__ void _GatherGradImpl(
    Tin* input,
    Tin* indices,
    T* grad_output,
    T* grad_weight,
    int64_t numel,
    int64_t input_numel,
    int64_t param_itrs,
    int64_t stride) {
  int idx = blockIdx.x * 4 + threadIdx.y;

  const int SZ = 4;
  if (idx < numel && (idx == 0 || input[idx] != input[idx - 1])) {
    do {
      for (int itr = 0; itr < param_itrs; ++itr) {
        const int start_feature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
        const int weight_row = itr * input_numel + ((int)input[idx]) * stride;  //the offset of the input
        const int grad_row = (itr * numel + ((int)indices[idx])) * stride;      //the offset of the gradient

        T gradient[SZ];
        T weight[SZ];

#pragma unroll
        for (int ii = 0; ii < SZ; ii++) {
          int feature_dim = start_feature + ii * WARP_SIZE;
          if (feature_dim < stride) {
            gradient[ii] = static_cast<T>(grad_output[grad_row + feature_dim]);
            weight[ii] = static_cast<T>(grad_weight[weight_row + feature_dim]);
          }
        }

#pragma unroll
        for (int ii = 0; ii < SZ; ii++) {
          weight[ii] += gradient[ii];
        }

#pragma unroll
        for (int ii = 0; ii < SZ; ii++) {
          int feature_dim = start_feature + ii * WARP_SIZE;
          if (feature_dim < stride) {
            grad_weight[weight_row + feature_dim] = static_cast<T>(weight[ii]);
          }
        }
      }
      idx++;
    } while (idx < numel && input[idx] == input[idx - 1]);
  }
}

template <typename T>
__host__ __device__ __forceinline__ T CeilDiv(T a, T b) {
  return (a + b - 1) / b;
}

template <typename T, typename Tin>
void GatherGradImpl(
    const T* grad_data,
    const Tin* indices_data,
    const int64_t num_indices,
    const int64_t num_weights,
    const int64_t stride,
    Tin* origin_indices,
    T* output_data,
    const int64_t num_inputs,  //The number of input elements starting from the gathering dimension
    const int64_t param_itrs,  //The size of dimensions of the data before gathering dimension
    ThrustAllocator& allocator) {
  // sort the index
  auto count_iter = thrust::counting_iterator<Tin>(0);
  auto origin_data = thrust::device_ptr<Tin>(origin_indices);

  auto policy = thrust::hip::par(allocator);
  thrust::copy(policy, count_iter, count_iter + num_indices, origin_data);
  //TODO: remove the const_cast
  auto sorted_data = thrust::device_ptr<Tin>(const_cast<Tin*>(indices_data));
  thrust::sort_by_key(policy, sorted_data, sorted_data + num_indices, origin_data, thrust::less<Tin>());

  dim3 grid(CeilDiv(num_indices, (int64_t)4), CeilDiv(stride, (int64_t)128));
  dim3 block(WARP_SIZE, 4);

  hipLaunchKernelGGL(_GatherGradImpl, dim3(grid), dim3(block), 0, 0, 
      const_cast<Tin*>(indices_data),
      origin_indices,
      const_cast<T*>(grad_data),
      output_data,
      num_indices,
      num_inputs,
      param_itrs,
      stride);
}

#define SPECIALIZED_GRAD_IMPL2(T)                                                                                      \
  template void GatherGradImpl<T, int64_t>(const T* grad_data, const int64_t* indices_data,                            \
                                           const int64_t num_indices, const int64_t num_weights, const int64_t stride, \
                                           int64_t* origin_indices, T* output_data, const int64_t num_inputs,          \
                                           const int64_t params_itrs, ThrustAllocator& allocator);                     \
  template void GatherGradImpl<T, int32_t>(const T* grad_data, const int32_t* indices_data,                            \
                                           const int64_t num_indices, const int64_t num_weights, const int64_t stride, \
                                           int32_t* origin_indices, T* output_data, const int64_t num_inputs,          \
                                           const int64_t params_itrs, ThrustAllocator& allocator);

SPECIALIZED_GRAD_IMPL2(float)
SPECIALIZED_GRAD_IMPL2(half)

}  // namespace hip
}  // namespace onnxruntime
