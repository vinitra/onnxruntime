#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/hip/hip_common.h"
#include "core/providers/hip/cu_inc/common.cuh"
#include "core/providers/hip/atomic/common.cuh"
#include "gradient_control.h"

namespace onnxruntime {
namespace hip {
template <typename T, typename T_GRAD>
__global__ void _AccumulateGradient(
    const T* gradient_buffer,
    const T_GRAD* gradient,
    T* accumulated_gradient,
    HIP_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  accumulated_gradient[id] = gradient_buffer[id] + T(gradient[id]);
}

template <typename T, typename T_GRAD>
void AccumulateGradientImpl(
    const T* gradient_buffer,
    const T_GRAD* gradient,
    T* accumulated_gradient,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  HIP_LONG N = static_cast<HIP_LONG>(count);
  hipLaunchKernelGGL(_AccumulateGradient<T, T_GRAD>, dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
      gradient_buffer,
      gradient,
      accumulated_gradient,
      N);
}

#define SPECIALIZED_IMPL_AccumulateGradient(T, T_GRAD) \
  template void AccumulateGradientImpl(                \
      const T* gradient_buffer,                        \
      const T_GRAD* gradient,                          \
      T* accumulated_gradient,                         \
      size_t count);

SPECIALIZED_IMPL_AccumulateGradient(float, float)
SPECIALIZED_IMPL_AccumulateGradient(float, half)

}  // namespace hip
}  // namespace onnxruntime