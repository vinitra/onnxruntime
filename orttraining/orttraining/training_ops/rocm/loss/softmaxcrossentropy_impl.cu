#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/rocm/cu_inc/common.cuh"
#include "softmaxcrossentropy_impl.h"
#include "core/providers/rocm/rocm_common.h"

namespace onnxruntime {
namespace rocm {

template <typename T>
__global__ void _SoftMaxCrossEntropy(
    const T* log_prob_data,
    const T* label_data,
    HIP_LONG NORMALIZE_FACTOR,
    T* output_data,
    HIP_LONG N) {

  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  output_data[id] = -log_prob_data[id] * label_data[id] / NORMALIZE_FACTOR;
}

template <typename T>
void SoftMaxCrossEntropyImpl(
    const T* log_prob,
    const T* label,
    size_t normalize_factor,
    T* output_data,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  HIP_LONG N = static_cast<HIP_LONG>(count);
  HIP_LONG NORMALIZE_FACTOR = static_cast<HIP_LONG>(normalize_factor);
  hipLaunchKernelGGL(HIP_KERNEL_NAME(_SoftMaxCrossEntropy<T>), dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
      log_prob,
      label,
      NORMALIZE_FACTOR,
      output_data,
      N);
}

#define SPECIALIZED_IMPL_SoftMaxEntropyImpl(T) \
  template void SoftMaxCrossEntropyImpl(       \
      const T* log_prob,                       \
      const T* label,                          \
      size_t normalize_factor,                 \
      T* output_data,                          \
      size_t count);

SPECIALIZED_IMPL_SoftMaxEntropyImpl(float)

template <typename T>
__global__ void _SoftMaxCrossEntropyGrad(
    const T* dY,
    const T* log_prob,
    const T* label,
    HIP_LONG NORMALIZE_FACTOR,
    T* output_data,
    HIP_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  output_data[id] = (_Exp(log_prob[id]) - label[id]) * (*dY) / NORMALIZE_FACTOR;
}

template <typename T>
void SoftMaxCrossEntropyGradImpl(
    const T* dY,
    const T* log_prob,
    const T* label,
    size_t normalize_factor,
    T* output_data,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  HIP_LONG N = static_cast<HIP_LONG>(count);
  HIP_LONG NORMALIZE_FACTOR = static_cast<HIP_LONG>(normalize_factor);
  hipLaunchKernelGGL(HIP_KERNEL_NAME(_SoftMaxCrossEntropyGrad<T>), dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
      dY,
      log_prob,
      label,
      NORMALIZE_FACTOR,
      output_data,
      N);
}

#define SPECIALIZED_IMPL_SoftMaxEntropyGradImpl(T) \
  template void SoftMaxCrossEntropyGradImpl(       \
      const T* dY,                                 \
      const T* log_prob,                           \
      const T* label,                              \
      size_t normalize_factor,                     \
      T* output_data,                              \
      size_t count);

SPECIALIZED_IMPL_SoftMaxEntropyGradImpl(float)

template <typename T, typename Tin>
__global__ void _SparseSoftmaxCrossEntropy(
    const T* log_prob_data,
    const Tin* label_data,
    const T* normalize_factor_data,
    T* output_data,
    HIP_LONG N,
    HIP_LONG D) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(i, N);
  HIP_KERNEL_ASSERT(label_data[i] >= 0 && label_data[i] < D);
  if (*normalize_factor_data == 0) {
    output_data[i] = 0;
  } else {
    output_data[i] = -log_prob_data[i * D + label_data[i]] / (*normalize_factor_data);
  }
}

template <typename T, typename Tin>
__global__ void _WeightedSparseSoftmaxCrossEntropy(
    const T* log_prob_data,
    const Tin* label_data,
    const T* weight_data,
    const T* normalize_factor_data,
    T* output_data,
    HIP_LONG N,
    HIP_LONG D) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(i, N);
  HIP_KERNEL_ASSERT(label_data[i] >= 0 && label_data[i] < D);
  if (*normalize_factor_data == 0) {
    output_data[i] = 0;
  } else {
    output_data[i] = -log_prob_data[i * D + label_data[i]] * weight_data[i] / (*normalize_factor_data);
  }
}

template <typename T, typename Tin>
void SparseSoftmaxCrossEntropyImpl(
    const T* log_prob,
    const Tin* label,
    const T* weight,
    const T* normalize_factor,
    T* output_data,
    size_t count,
    size_t label_depth) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  HIP_LONG N = static_cast<HIP_LONG>(count);
  HIP_LONG D = static_cast<HIP_LONG>(label_depth);
  if (weight) {
    hipLaunchKernelGGL(HIP_KERNEL_NAME(_WeightedSparseSoftmaxCrossEntropy<T, Tin>), dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
      log_prob,
      label,
      weight,
      normalize_factor,
      output_data,
      N,
      D);
  } else {
    hipLaunchKernelGGL(HIP_KERNEL_NAME(_SparseSoftmaxCrossEntropy<T, Tin>), dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
        log_prob,
        label,
        normalize_factor,
        output_data,
        N,
        D);
  }
}

#define SPECIALIZED_IMPL_SparseSoftMaxEntropyImpl(T, Tin) \
  template void SparseSoftmaxCrossEntropyImpl(            \
      const T* log_prob,                                  \
      const Tin* label,                                   \
      const T* weight,                                    \
      const T* normalize_factor,                          \
      T* output_data,                                     \
      size_t count,                                       \
      size_t label_depth);

SPECIALIZED_IMPL_SparseSoftMaxEntropyImpl(float, int32_t)
SPECIALIZED_IMPL_SparseSoftMaxEntropyImpl(float, int64_t)

template <typename T, typename Tin>
__global__ void _SparseSoftmaxCrossEntropyGrad(
    const T* dY,
    const T* log_prob,
    const Tin* label,
    const T* normalize_factor,
    T* output_data,
    HIP_LONG N,
    HIP_LONG D) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(i, N * D);
  int row = i / D;
  int d = i % D;
  if (*normalize_factor == 0) {
    output_data[i] = 0;
  } else {
    output_data[i] = (*dY) * (_Exp(log_prob[i]) - 1.0 * (d == label[row])) / (*normalize_factor);
  }
}

template <typename T, typename Tin>
__global__ void _WeightedSparseSoftmaxCrossEntropyGrad(
    const T* dY,
    const T* log_prob,
    const Tin* label,
    const T* weight,
    const T* normalize_factor,
    T* output_data,
    HIP_LONG N,
    HIP_LONG D) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(i, N * D);
  int row = i / D;
  int d = i % D;
  if (*normalize_factor == 0) {
    output_data[i] = 0;
  } else {
    output_data[i] = (*dY) * weight[row] * (_Exp(log_prob[i]) - 1.0 * (d == label[row])) / (*normalize_factor);
  }
}

template <typename T, typename Tin>
void SparseSoftmaxCrossEntropyGradImpl(
    const T* dY,
    const T* log_prob,
    const Tin* label,
    const T* weight,
    const T* normalize_factor,
    T* output_data,
    size_t count,
    size_t label_depth) {
  HIP_LONG N = static_cast<HIP_LONG>(count);
  HIP_LONG D = static_cast<HIP_LONG>(label_depth);
  int blocksPerGrid = (int)(ceil(static_cast<float>(N * D) / GridDim::maxThreadsPerBlock));
  if (weight) {
    hipLaunchKernelGGL(HIP_KERNEL_NAME(_WeightedSparseSoftmaxCrossEntropyGrad<T, Tin>), dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
      dY,
      log_prob,
      label,
      weight,
      normalize_factor,
      output_data,
      N,
      D);
  } else {
    hipLaunchKernelGGL(HIP_KERNEL_NAME(_SparseSoftmaxCrossEntropyGrad<T, Tin>), dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
        dY,
        log_prob,
        label,
        normalize_factor,
        output_data,
        N,
        D);
  }
}

#define SPECIALIZED_IMPL_SparseSoftMaxEntropyGradImpl(T, Tin) \
  template void SparseSoftmaxCrossEntropyGradImpl(            \
      const T* dY,                                            \
      const T* log_prob,                                      \
      const Tin* label,                                       \
      const T* weight,                                        \
      const T* normalize_factor,                              \
      T* output_data,                                         \
      size_t count,                                           \
      size_t label_depth);

SPECIALIZED_IMPL_SparseSoftMaxEntropyGradImpl(float, int32_t)
SPECIALIZED_IMPL_SparseSoftMaxEntropyGradImpl(float, int64_t)

}  // namespace rocm
}  // namespace onnxruntime
